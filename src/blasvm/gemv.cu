#include <stdio.h>
#include <hip/hip_runtime.h>
#include <typeinfo>

// Templated CUDA kernel for GEMV: y = alpha * A * x + beta * y
template <typename T>
__global__ void gemv_kernel(T *A, T *x, T *y, T alpha, T beta, int rows, int cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows) {
        T result = 0;
        for (int col = 0; col < cols; col++) {
            result += A[row * cols + col] * x[col];
        }
        y[row] = alpha * result + beta * y[row];
    }
}

// Templated GEMV function with FLOPS and Bandwidth calculation
template <typename T>
void gemv(const T *h_A, const T *h_x, T *h_y, T alpha, T beta, int rows, int cols) {
    // Calculate sizes
    int size_A = rows * cols * sizeof(T);
    int size_x = cols * sizeof(T);
    int size_y = rows * sizeof(T);

    // Device memory pointers
    T *d_A, *d_x, *d_y;

    // Allocate memory on the device (GPU)
    hipMalloc((void **)&d_A, size_A);
    hipMalloc((void **)&d_x, size_x);
    hipMalloc((void **)&d_y, size_y);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, size_x, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, size_y, hipMemcpyHostToDevice);

    // Define thread block and grid size
    int blockSize = 256;
    int gridSize = (rows + blockSize - 1) / blockSize;

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start timing
    hipEventRecord(start, 0);

    // Launch the kernel
    gemv_kernel<<<gridSize, blockSize>>>(d_A, d_x, d_y, alpha, beta, rows, cols);

    // Stop timing
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Calculate FLOPS
    size_t flops = 2 * static_cast<size_t>(rows) * static_cast<size_t>(cols); // 2 operations (mul + add) per element
    double gflops = (flops / 1e9) / (milliseconds / 1000.0);  // GFLOPS = flops / 10^9

    // Calculate memory bandwidth
    size_t mem_access = (rows * cols + cols + rows) * sizeof(T);  // Total memory accessed
    double bandwidth = (mem_access / 1e9) / (milliseconds / 1000.0);  // Bandwidth in GB/s

    // Output the results
    printf("Time for GEMV (DataType: %s): %f ms\n", typeid(T).name(), milliseconds);
    printf("Performance: %f GFLOPS\n", gflops);
    printf("Memory Bandwidth: %f GB/s\n", bandwidth);

    // Copy the result back to host
    hipMemcpy(h_y, d_y, size_y, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}