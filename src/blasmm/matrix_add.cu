#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA Kernel function to add elements of two arrays
__global__ void matrix_add_ker(int *a, int *b, int *c, int width, int height) {
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int col = threadIdx.y + blockIdx.y * blockDim.y;
    int index = row * width + col;
    if (row < height && col < width) {
        c[index] = a[index] + b[index];
    }
}

void matrix_add(const int* h_a, const int* h_b, int* h_c, int width, int height) {
    // int width = 200;
    // int height = 300; // Size of arrays
    int n = width * height;
    int size = n * sizeof(int);

    // Device copies of a, b, c
    int *d_a, *d_b, *d_c;

    // Allocate space for host copies of a, b, c and setup input values
    // h_a = (int *)malloc(size);
    // h_b = (int *)malloc(size);
    // h_c = (int *)malloc(size);

    // for (int row = 0; row < height; row++) {
    //     for (int col = 0; col < width; col++) {
    //         h_a[row * width + col] = row + col;
    //         h_b[row * width + col] = row - col;
    //     }
    // }

    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy inputs to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Launch add() kernel on the GPU
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (height + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    // Time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
   
    matrix_add_ker<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, width, height);

    // Time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Performance metrics
    // 1. Time
    printf("Time for matrix addition: %f ms\n", milliseconds);

    // 2. FLOPS (Matrix Add: 1 addition per element)
    int flops = width * height;  // Each element involves 1 addition
    float flops_per_second = (flops / (milliseconds / 1000.0f)) / 1e9;  // Convert to GFLOPS
    printf("Performance: %f GFLOPS\n", flops_per_second);

    // 3. Bandwidth (Memory bound calculation)
    // We are reading two matrices (a, b) and writing one matrix (c)
    float bandwidth = (3.0f * size / (milliseconds / 1000.0f)) / (1 << 30);  // GB/s
    printf("Memory Bandwidth: %f GB/s\n", bandwidth);

    // Copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Print out some results
    // Print result
    // int h = 20 < height? 20:height;
    // int w = 20 < width? 20:width;
    // printf("%d,%d", h, w);
    // printf("Matrix A:\n");
    // for (int row = 0; row < h; row++) {
    //     for (int col = 0; col < w; col++) {
    //         printf("%d ", h_a[row * width + col]);
    //     }
    //     printf("\n");
    // }

    // printf("\nMatrix B:\n");
    // for (int row = 0; row < h; row++) {
    //     for (int col = 0; col < w; col++) {
    //         printf("%d ", h_b[row * width + col]);
    //     }
    //     printf("\n");
    // }

    // printf("\nResult (Matrix C = A + B):\n");
    // for (int row = 0; row < h; row++) {
    //     for (int col = 0; col < w; col++) {
    //         printf("%d ", h_c[row * width + col]);
    //     }
    //     printf("\n");
    // }

    // Cleanup
    // free(h_a);
    // free(h_b);
    // free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return;
}
