#include <hip/hip_runtime.h>
#include <stdio.h>
#include <typeinfo>

// Templated CUDA kernel for GEMM: C = alpha * A * B + beta * C
template <typename T>
__global__ void gemm_kernel(T *A, T *B, T *C, T alpha, T beta, int A_rows, int A_cols, int B_cols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < A_rows && col < B_cols) {
        T value = 0;
        for (int k = 0; k < A_cols; k++) {
            value += A[row * A_cols + k] * B[k * B_cols + col];
        }
        C[row * B_cols + col] = alpha * value + beta * C[row * B_cols + col];
    }
}

// Templated GEMM function
template <typename T>
void gemm(const T *h_A, const T *h_B, T *h_C, T alpha, T beta, int A_rows, int A_cols, int B_cols) {
    // Calculate sizes
    int size_A = A_rows * A_cols * sizeof(T);
    int size_B = A_cols * B_cols * sizeof(T);
    int size_C = A_rows * B_cols * sizeof(T);

    // Device memory pointers
    T *d_A, *d_B, *d_C;
    printf("here\n");

    // Allocate memory on the device (GPU)
    hipMalloc((void **)&d_A, size_A);
    hipMalloc((void **)&d_B, size_B);
    hipMalloc((void **)&d_C, size_C);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, size_C, hipMemcpyHostToDevice);

    // Define thread block and grid size
    dim3 blockSize(16, 16);
    dim3 gridSize((B_cols + blockSize.x - 1) / blockSize.x, (A_rows + blockSize.y - 1) / blockSize.y);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start timing
    hipEventRecord(start, 0);

    // Launch the kernel
    gemm_kernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, alpha, beta, A_rows, A_cols, B_cols);

    // Stop timing
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time for GEMM (DataType: %s): %f ms\n", typeid(T).name(), milliseconds);

    // Copy the result back to host
    hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

template void gemm<float>(const float *h_A, const float *h_B, float *h_C, float alpha, float beta, int A_rows, int A_cols, int B_cols);
template void gemm<double>(const double *h_A, const double *h_B, double *h_C, double alpha, double beta, int A_rows, int A_cols, int B_cols);