#include <hip/hip_runtime.h>
#include <stdio.h>
#include <typeinfo>

// CUDA error checking macro
#define CHECK_CUDA(call) {                                          \
    hipError_t err = (call);                                       \
    if (err != hipSuccess) {                                       \
        fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n",\
                __FILE__, __LINE__, hipGetErrorString(err));       \
        exit(EXIT_FAILURE);                                         \
    }                                                               \
}

// Templated CUDA kernel for GEMM: C = alpha * A * B + beta * C
__global__ void gemm_kernel(float *A, float *B, float *C, float alpha, float beta, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < k) {
        float value = 0;
        for (int j = 0; j < n; j++) {
            value += A[row * n + j] * B[j * k + col];
        }
        C[row * k + col] = alpha * value + beta * C[row * k + col];
    }
}

// Templated GEMM function
void gemm(const float *h_A, const float *h_B, float *h_C, float alpha, float beta, int m, int n, int k) {
    // Calculate sizes
    int size_A = m * n * sizeof(float);
    int size_B = n * k * sizeof(float);
    int size_C = m * k * sizeof(float);

    // Device memory pointers
    float *d_A, *d_B, *d_C;

    // Allocate memory on the device (GPU)
    CHECK_CUDA(hipMalloc((void **)&d_A, m * n * sizeof(float)));
    CHECK_CUDA(hipMalloc((void **)&d_B, size_B));
    CHECK_CUDA(hipMalloc((void **)&d_C, size_C));

    // Copy data from host to device
    CHECK_CUDA(hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_C, h_C, size_C, hipMemcpyHostToDevice));

    // Define thread block and grid size
    dim3 blockSize(16, 16);
    dim3 gridSize((k + blockSize.x - 1) / blockSize.x, (m + blockSize.y - 1) / blockSize.y);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Timing and launch the kernel
    hipEventRecord(start);    
    gemm_kernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, alpha, beta, m, n, k);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float time_ms = 0.f;
    hipEventElapsedTime(&time_ms, start, stop);
    printf("%f ms used.\n", time_ms);
    long ops = (long)m * n * k * 2;
    double gops = ((double)ops / 1e9) / ((double)time_ms / 1e3);
    printf("My GEMM: %f Gops\n", gops);

    // Copy the result back to host
    CHECK_CUDA(hipMemcpy(h_C, d_C, size_C, hipMemcpyDeviceToHost));

    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}