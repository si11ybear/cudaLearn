
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA Kernel function to add elements of two arrays
__global__ void add(int *a, int *b, int *c, int width, int height) {
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int col = threadIdx.y + blockIdx.y * blockDim.y;
    int index = row * width + col;
    if (row < height && col < width) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    int width = 200;
    int height = 300; // Size of arrays
    int n = width * height;
    int size = n * sizeof(int);

    // Host copies of a, b, c
    int *h_a, *h_b, *h_c;

    // Device copies of a, b, c
    int *d_a, *d_b, *d_c;

    // Allocate space for host copies of a, b, c and setup input values
    h_a = (int *)malloc(size);
    h_b = (int *)malloc(size);
    h_c = (int *)malloc(size);

    for (int row = 0; row < height; row++) {
        for (int col = 0; col < width; col++) {
            h_a[row * width + col] = row + col;
            h_b[row * width + col] = row - col;
        }
    }

    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy inputs to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Launch add() kernel on the GPU
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((width + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (height + threadsPerBlock.y - 1) / threadsPerBlock.y);
    add<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, width, height);

    // Copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Print out some results
    // Print result
    int h = 20 < height? 20:height;
    int w = 20 < width? 20:width;
    printf("%d,%d", h, w);
    printf("Matrix A:\n");
    for (int row = 0; row < h; row++) {
        for (int col = 0; col < w; col++) {
            printf("%d ", h_a[row * width + col]);
        }
        printf("\n");
    }

    printf("\nMatrix B:\n");
    for (int row = 0; row < h; row++) {
        for (int col = 0; col < w; col++) {
            printf("%d ", h_b[row * width + col]);
        }
        printf("\n");
    }

    printf("\nResult (Matrix C = A + B):\n");
    for (int row = 0; row < h; row++) {
        for (int col = 0; col < w; col++) {
            printf("%d ", h_c[row * width + col]);
        }
        printf("\n");
    }

    // Cleanup
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
